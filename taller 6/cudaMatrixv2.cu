#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
//#define rows 1000
//#define cols 1000
using namespace std;

// CUDA kernel. Each thread takes care of one element of c
__global__ void matricesMul(double *m1, double *m2, double *m3){
		// Get our global thread ID
		int ti = blockIdx.y*blockDim.y+threadIdx.y;
		int tj = blockIdx.x*blockDim.x+threadIdx.x;
		// Make sure we do not go out of bounds
		if(ti < rows && tj < cols){
			double data= 0.0;
			for(int k=0;k<rows;k++) data += m1[ti*rows+k] * m2[k*cols+tj];
			m3[ti*rows+tj] = data;
		}
}

FILE * openFile(char const *fileName){
	/* try to open a file */
	FILE *f=NULL;
	f = fopen(fileName,"r");
	if(f == NULL){
		printf("File '%s' doesn't exist!\n",fileName);
		exit(1);
	}
	return f;
}

double * buildMatrix(FILE *f, size_t &rows, size_t &columns){
	/* build a matrix M (get memory) */
	fscanf(f,"%i",&rows); /* %zu zx is size_t */
	fscanf(f,"%i",&columns);
	//fgetc(f);  /* skipping nasty character, on this case new line */
	double *M = (double *)malloc(rows*columns*sizeof(double)); /* reserved memory */
	if(rows <= 0 || columns <= 0){
		printf("should be size positive and upper than cero \n");
		exit(1);
	}
	return M;
}

void getData(FILE *f, double *M, size_t len){
	/* Capture data from plain text file to system memory
	Note: the data files need one end line to get last number
	format of data files 
	...
	A
	B
	#.#,#.#
	#.#,#.#

	...
	A -> Size row
	B -> Size column
	*/
	//sizeof(char)==1
	char data[50]="", ch = ' ';
	size_t posData = 0, Mindex = 0;
	while(len>Mindex){
		ch = fgetc(f); /*get char and char in file f */
		if(Mindex==0 && ch == '\n'){//skip nasty chracter
			continue;
		}
		if(ch == ',' || ch == '\n'){
			data[posData] = '\0'; /* char end */
			M[Mindex] = stod(data); /*convert string to double */
			posData = 0;
			strcpy(data, ""); /* take memory for the next data */
			Mindex++;
		}else{
			data[posData] = ch;
			posData++;
		}

	}
}
void writeResult(){
	/*
		Wite the result on output.txt file
		M -> Matrix, Mrow -> Matrix rows, Mcol -> Matrix columns
	*/
	FILE *f = fopen("output.txt","w+");//clean file and set result
	for(size_t i=0;i<this->M1row;i++){
		for(size_t j=0;j<this->M2col;j++){
			if(j+1 == this->M2col) {//last chracter
				fprintf(f,"%f\n",this->MResult[i*this->M2col + j]);
			}
			else {
				fprintf(f,"%f,",this->MResult[i*this->M2col + j]);
			}
		}
	}
	fclose(f);
}

bool checkMul(){
	if(this->M1col != this->M2row){
		printf("ERROR - Matrices cannot be multiply!"); 
		return 0;//FALSE
	}
	return 1;//TRUE
}

int main( int argc, char* argv[] ){
	if(argc != 2){
		printf("There should be 2 arguments!\n");
		exit(1);
	}
	// Host (CPU) input matrices
	double *h_m1;
	size_t rows_m1, cols_m1;
	double *h_m2;
	size_t rows_m2, cols_m2;
	//Host (CPU) output matrix
	double *h_m3;

	// Device (GPU-Nvidia) input matrices
	double *d_m1;
	double *d_m2;
	//Device (GPU-Nvidia) output matrix
	double *d_m3;

	FILE *f1=NULL; /* file pointers */
	f1=openFile(argv[1]);
    // Allocate memory for each matrix on host
	h_m1=buildMatrix(f1, rows_m1, cols_m1);
	
	getData(f1, h_m1, rows_m1*cols_m1);

	h_m2=buildMatrix(f1, rows_m2, cols_m2);
	getData(f1, h_m2, rows_m2*cols_m2);

	h_m3=(double *)malloc(rows_m1*cols_m2*sizeof(double));
	
	fclose(f1);
	// Size of matrices n²
    size_t n = rows_m1*cols_m2;
    size_t bytes = n*sizeof(double);

	// Allocate memory for each matrix on GPU
	hipMalloc((void **)&d_m1, bytes);
	hipMalloc((void **)&d_m2, bytes);
	hipMalloc((void **)&d_m3, bytes);


	// Copy host matrices to device
	hipMemcpy( d_m1, h_m1, bytes, hipMemcpyHostToDevice);
	hipMemcpy( d_m2, h_m2, bytes, hipMemcpyHostToDevice);

	// Number of threads in each thread matrix block
	double x = sqrt(1024);
	size_t threadsInX= floor(x);
	size_t threadsInY= threadsInX;
	dim3 dimBlock(threadsInX,threadsInY,1);
	// Number of thread blocks in matrix grid
	size_t gridNum = ceil((double)n/1024);  // needed grid numbers to our problem
	size_t gridR = ceil(sqrt(gridNum)); 		// grid rows	
	size_t gridC = gridR;										// grid cols
	dim3 dimGrid(gridR,gridC,1);

	// Execute the kernel
	matricesMul<<<dimGrid,dimBlock>>>(d_m1, d_m2, d_m3);

	// Copy result m3 matrix back to host
	hipMemcpy(h_m3, d_m3, bytes, hipMemcpyDeviceToHost);

	// print every item into m3 matrix
	for(int i=0; i<n; i++){
		double val = h_m3[i];
		printf("final result: %f\n", val);
	}

	// Release device memory
	hipFree(d_m1);
	hipFree(d_m2);
	hipFree(d_m3);

	// Release host memory
	free(h_m1);
	free(h_m2);
	free(h_m3);

	return 0;
}
