
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define rows 1000
#define cols 1000

// CUDA kernel. Each thread takes care of one element of c
__global__ void matricesMul(double *m1, double *m2, double *m3)
{
    // Get our global thread ID
    int ti = blockIdx.y*blockDim.y+threadIdx.y;
    int tj = blockIdx.x*blockDim.x+threadIdx.x;
    // Make sure we do not go out of bounds
    if(ti < rows && tj < cols){
			double data= 0.0;
      for(int k=0;k<rows;k++) data += m1[ti*rows+k] * m2[k*cols+tj];
      m3[ti*rows+tj] = data;
    }
}

int main( int argc, char* argv[] ){
    // Size of matrices n²
    int n = rows*cols;

    // Host input matrices
    double *h_m1;
    double *h_m2;
    //Host output matrix
    double *h_m3;

    // Device input matrices
    double *d_m1;
    double *d_m2;
    //Device output matrix
    double *d_m3;

    // Size, in bytes, of each matrix
    size_t bytes = n*sizeof(double);

    // Allocate memory for each matrix on host
    h_m1 = (double*)malloc(bytes);
    h_m2 = (double*)malloc(bytes);
    h_m3 = (double*)malloc(bytes);

    // Allocate memory for each matrix on GPU
    hipMalloc((void **)&d_m1, bytes);
    hipMalloc((void **)&d_m2, bytes);
    hipMalloc((void **)&d_m3, bytes);

    // Initialize matrices on host
    for(int i=0; i<n; i++){
      h_m1[i] = sin(i)*sin(i);
      h_m2[i] = cos(i)*cos(i);
    }

    // Copy host matrices to device
    hipMemcpy( d_m1, h_m1, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_m2, h_m2, bytes, hipMemcpyHostToDevice);

    // Number of threads in each thread matrix block
		double x = sqrt(1024);
		size_t threadsInX= floor(x);
		size_t threadsInY= threadsInX;
    dim3 dimBlock(threadsInX,threadsInY,1);
    // Number of thread blocks in matrix grid
		size_t gridNum = ceil((double)n/1024);  // needed grid numbers to our problem
		size_t gridR = ceil(sqrt(gridNum)); 		// grid rows	
		size_t gridC = gridR;										// grid cols
    dim3 dimGrid(gridR,gridC,1);

    // Execute the kernel
    matricesMul<<<dimGrid,dimBlock>>>(d_m1, d_m2, d_m3);

    // Copy result m3 matrix back to host
    hipMemcpy(h_m3, d_m3, bytes, hipMemcpyDeviceToHost);

    // print every item into m3 matrix
    for(int i=0; i<n; i++){
			double val = h_m3[i];
			printf("final result: %f\n", val);
		}

    // Release device memory
    hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_m3);

    // Release host memory
    free(h_m1);
    free(h_m2);
    free(h_m3);

    return 0;
}
